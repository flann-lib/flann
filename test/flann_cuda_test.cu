#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <time.h>
#define FLANN_USE_CUDA
#include <flann/flann.h>
#include <flann/io/hdf5.h>
#include <flann/nn/ground_truth.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector_functions.h>

using namespace flann;


float compute_precision(const flann::Matrix<int>& match, const flann::Matrix<int>& indices)
{
    int count = 0;

    assert(match.rows == indices.rows);
    size_t nn = std::min(match.cols, indices.cols);

    for (size_t i=0; i<match.rows; ++i) {
        for (size_t j=0;j<nn;++j) {
            for (size_t k=0;k<nn;++k) {
                if (match[i][j]==indices[i][k]) {
                    count ++;
                }
                else
				{
// 					std::cout<<i<<":"<<match[i][j]<<"!="<<indices[i][k]<<std::endl;
				}
            }
        }
    }

    return float(count)/(nn*match.rows);
}

struct smallerWithTolerance
{
	float tol;
	bool operator()(float a, float b )
	{
		return a<(b-tol);
	}
};

/** @brief Compare the distances for match accuracies
 * This is more precise: e.g. when you ask for the top 10 neighbors and they all get the same distance,
 * you might have 100 other neighbors that are at the same distance and simply matching the indices is not the way to go
 * @param gt_dists the ground truth best distances
 * @param dists the distances of the computed nearest neighbors
 * @param tol tolerance at which distanceare considered equal
 * @return
 */
template<typename T>
float computePrecisionDiscrete(const flann::Matrix<T>& gt_dists, const flann::Matrix<T>& dists, float tol)
{
  int count = 0;

  assert(gt_dists.rows == dists.rows);
  size_t nn = std::min(gt_dists.cols, dists.cols);
  std::vector<T> gt_sorted_dists(nn), sorted_dists(nn), intersection(nn);

  smallerWithTolerance swt;
  swt.tol=tol;
  for (size_t i = 0; i < gt_dists.rows; ++i)
  {
    std::copy(gt_dists[i], gt_dists[i] + nn, gt_sorted_dists.begin());
    std::sort(gt_sorted_dists.begin(), gt_sorted_dists.end());
    std::copy(dists[i], dists[i] + nn, sorted_dists.begin());
    std::sort(sorted_dists.begin(), sorted_dists.end());
    typename std::vector<T>::iterator end = std::set_intersection(gt_sorted_dists.begin(), gt_sorted_dists.end(),
                                                             sorted_dists.begin(), sorted_dists.end(),
                                                             intersection.begin(),swt);
    count += (end - intersection.begin());
  }

  return float(count) / (nn * gt_dists.rows);
}

class FLANNTestFixture : public ::testing::Test {
protected:
    clock_t start_time_;

    void start_timer(const std::string& message = "")
    {
        if (!message.empty()) {
            printf("%s", message.c_str());
            fflush(stdout);
        }
        start_time_ = clock();
    }

    double stop_timer()
    {
        return double(clock()-start_time_)/CLOCKS_PER_SEC;
    }

};



class Flann_3D : public FLANNTestFixture {
protected:
    flann::Matrix<float> data;
    flann::Matrix<float> query;
    flann::Matrix<int> match;
    flann::Matrix<float> dists;
    flann::Matrix<int> indices;

    void SetUp()
    {
        printf("Reading test data...");
        fflush(stdout);
        flann::load_from_file(data, "cloud.h5","dataset");
        flann::load_from_file(query,"cloud.h5","query");
        flann::load_from_file(match,"cloud.h5","indices");

        dists = flann::Matrix<float>(new float[query.rows*5], query.rows, 5);
        indices = flann::Matrix<int>(new int[query.rows*5], query.rows, 5);
        printf("done\n");
    }

    void TearDown()
    {
        delete[] data.ptr();
        delete[] query.ptr();
        delete[] match.ptr();
        delete[] dists.ptr();
        delete[] indices.ptr();
    }
};


TEST_F(Flann_3D, KDTreeSingleTest)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeSingleIndexParams(12, false));
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

    start_timer("Searching KNN...");
    index.knnSearch(query, indices, dists, 5, flann::SearchParams(-1) );
    printf("done (%g seconds)\n", stop_timer());

    float precision = compute_precision(match, indices);
    EXPECT_GE(precision, 0.99);
    printf("Precision: %g\n", precision);
}


TEST_F(Flann_3D, KDTreeCudaTest)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

    start_timer("Searching KNN...");
    index.knnSearch(query, indices, dists, 5, flann::SearchParams(-1) );
    printf("done (%g seconds)\n", stop_timer());

    float precision = compute_precision(match, indices);
    EXPECT_GE(precision, 0.99);
    printf("Precision: %g\n", precision);
}


class Flann_3D_Random_Cloud : public FLANNTestFixture {
protected:
    flann::Matrix<float> data;
    flann::Matrix<float> query;
    flann::Matrix<float> dists;
    flann::Matrix<int> indices;
	flann::Matrix<float> gt_dists;
	flann::Matrix<int> gt_indices;

    void SetUp()
    {
		const int n_points=10000;
		printf("creating random point cloud (%d points)...", n_points);
		data = flann::Matrix<float>(new float[n_points*3], n_points, 3);
		srand(1);
		for( int i=0; i<n_points; i++ )
		{
			data[i][0]=rand()/float(RAND_MAX);
			data[i][1]=rand()/float(RAND_MAX);
			data[i][2]=rand()/float(RAND_MAX);
// 			std::cout<<data[i][0]<<" "<<data[i][1]<<" "<<data[i][2]<<std::endl;
		}
		
		query= flann::Matrix<float>(new float[n_points*3], n_points, 3);
		for( int i=0; i<n_points; i++ )
		{
			query[i][0]=data[i][0];//float(rand())/RAND_MAX;
			query[i][1]=data[i][1];//float(rand())/RAND_MAX;
			query[i][2]=data[i][2];//float(rand())/RAND_MAX;
// 			std::cout<<query[i][0]<<" "<<query[i][1]<<" "<<query[i][2]<<std::endl;
		}
		
		
        printf("done\n");
		
		const int max_nn = 16;
		
        dists = flann::Matrix<float>(new float[query.rows*max_nn], query.rows, max_nn);
		gt_dists = flann::Matrix<float>(new float[query.rows*max_nn], query.rows, max_nn);
        indices = flann::Matrix<int>(new int[query.rows*max_nn], query.rows, max_nn);
		gt_indices = flann::Matrix<int>(new int[query.rows*max_nn], query.rows, max_nn);
		
		
		Index<L2<float> > index(data, flann::LinearIndexParams());
		start_timer("Building linear index...");
		index.buildIndex();
		printf("done (%g seconds)\n", stop_timer());
		
		start_timer("Searching KNN...");
		index.knnSearch(data, gt_indices, gt_dists, max_nn, flann::SearchParams() );
// 		for( int i=0; i<gt_dists.rows; i++ )
// 		{
// 			std::cout<<gt_indices[i][0]<<" "<<gt_dists[i][0]<<std::endl;
// 		}
		printf("done (%g seconds)\n", stop_timer());
    }

    void TearDown()
    {
        delete[] data.ptr();
        delete[] query.ptr();
        delete[] dists.ptr();
		delete[] gt_dists.ptr();
        delete[] indices.ptr();
		delete[] gt_indices.ptr();
		
    }
};

TEST_F(Flann_3D_Random_Cloud, Test1NN)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

    start_timer("Searching KNN...");
	indices.cols=1;
	dists.cols=1;
    index.knnSearch(query, indices, dists, 1, flann::SearchParams() );
    printf("done (%g seconds)\n", stop_timer());

//     float precision = compute_precision(gt_indices,indices);
	float precision = computePrecisionDiscrete(gt_dists,dists, 0);
    EXPECT_GE(precision, 0.99);
    printf("Precision: %g\n", precision);
	
}

TEST_F(Flann_3D_Random_Cloud, Test4NN)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

    start_timer("Searching KNN...");
	indices.cols=4;
	dists.cols=4;
    index.knnSearch(query, indices, dists, 4, flann::SearchParams() );
    printf("done (%g seconds)\n", stop_timer());

//     float precision = compute_precision(gt_indices,indices);
	float precision = computePrecisionDiscrete(gt_dists,dists, 1e-08);
    EXPECT_GE(precision, 0.99);
    printf("Precision: %g\n", precision);
	
}

TEST_F(Flann_3D_Random_Cloud, Test4NNGpuBuffers)
{
	thrust::host_vector<float4> data_host(data.rows);
	for( int i=0; i<data.rows; i++ )
	{
		data_host[i]=make_float4(data[i][0],data[i][1],data[i][2],0);
	}
	thrust::device_vector<float4> data_device = data_host;
	thrust::host_vector<float4> query_host(data.rows);
	for( int i=0; i<data.rows; i++ )
	{
		query_host[i]=make_float4(query[i][0],query[i][1],query[i][2],0);
	}
	thrust::device_vector<float4> query_device = query_host;
	
	flann::Matrix<float> data_device_matrix( (float*)thrust::raw_pointer_cast(&data_device[0]),data.rows,3,4*4);
	flann::Matrix<float> query_device_matrix( (float*)thrust::raw_pointer_cast(&query_device[0]),data.rows,3,4*4);
	
	flann::KDTreeCuda3dIndexParams index_params;
	index_params["input_is_gpu_float4"]=true;
	flann::Index<L2_Simple<float> > index(data_device_matrix, index_params);
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());

	
	thrust::device_vector<int> indices_device(query.rows*4);
	thrust::device_vector<float> dists_device(query.rows*4);
	flann::Matrix<int> indices_device_matrix( (int*)thrust::raw_pointer_cast(&indices_device[0]),query.rows,4);
	flann::Matrix<float> dists_device_matrix( (float*)thrust::raw_pointer_cast(&dists_device[0]),query.rows,4);
	
    start_timer("Searching KNN...");
	indices.cols=4;
	dists.cols=4;
	flann::SearchParams sp;
	sp.matrices_in_gpu_ram=true;
    index.knnSearch(query_device_matrix, indices_device_matrix, dists_device_matrix, 4, sp );
    printf("done (%g seconds)\n", stop_timer());
	
	flann::Matrix<int> indices_host( new int[ query.rows*4],query.rows,4 );
	flann::Matrix<float> dists_host( new float[ query.rows*4],query.rows,4 );
	
	thrust::copy( dists_device.begin(), dists_device.end(), dists_host.ptr() );
	thrust::copy( indices_device.begin(), indices_device.end(), indices_host.ptr() );

//     float precision = compute_precision(gt_indices,indices);
	float precision = computePrecisionDiscrete(gt_dists,dists_host, 1e-08);
    EXPECT_GE(precision, 0.99);
    printf("Precision: %g\n", precision);
	delete [] indices_host.ptr();
	delete [] dists_host.ptr();
}

TEST_F(Flann_3D_Random_Cloud, TestRadiusSearchVector)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());
	float r = 0.1;
	std::vector< std::vector<int> > indices;
	std::vector< std::vector<float> > dists;
	start_timer("Radius search, r=0.1");
	index.radiusSearch( query, indices,dists, r*r, flann::SearchParams() );
	printf("done (%g seconds)", stop_timer());

	start_timer("verifying results...");
	for( int i=0; i<query.rows; i++ )
	{
		for( int j=0; j<data.rows; j++ )
		{
			float dist = 0;
			for( int k=0; k<3; k++ )
				dist += (query[i][k]-data[j][k])*(query[i][k]-data[j][k]);
			if( dist < r*r )
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )!=indices[i].end() );
			}
			else
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )==indices[i].end() );
			}
		}
	}
	printf("done (%g seconds)\n", stop_timer());
	
	r=0.05;
	start_timer("Radius search, r=0.05");
	index.radiusSearch( query, indices,dists, r*r, flann::SearchParams() );
	printf("done (%g seconds)", stop_timer());
	
	start_timer("verifying results...");
	for( int i=0; i<query.rows; i++ )
	{
		for( int j=0; j<data.rows; j++ )
		{
			// for each pair of query and data points: either the distance between them
			// is smaller than r AND the point is in the result set, or 
			// the distance is larger and it is not.
			float dist = 0;
			for( int k=0; k<3; k++ )
				dist += (query[i][k]-data[j][k])*(query[i][k]-data[j][k]);
			if( dist < r*r )
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )!=indices[i].end() );
			}
			else
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )==indices[i].end() );
			}
		}
	}
	printf("done (%g seconds)\n", stop_timer());
}

TEST_F(Flann_3D_Random_Cloud, TestRadiusSearchMatrix)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());
	float r = 0.05;
	flann::Matrix<int> counts( new int[query.rows], query.rows,1);
	flann::Matrix<float> dummy( 0,0,0 );
	flann::SearchParams counting_params;
	counting_params.max_neighbors=0;
	start_timer("counting neighbors...");
	index.radiusSearch( query, counts,dummy, r*r, counting_params );
	printf("done (%g seconds)", stop_timer());
	
	int max_neighbors=0;
	for( int i=0; i<query.rows; i++ )
	{
		max_neighbors = std::max(max_neighbors, counts[i][0]);
	}
	EXPECT_TRUE(max_neighbors > 0 );
	flann::Matrix<int> indices( new int[max_neighbors*query.rows], query.rows, max_neighbors );
	flann::Matrix<float> dists( new float[max_neighbors*query.rows], query.rows, max_neighbors );
		
	start_timer("Radius search, r=0.05");
	index.radiusSearch( query, indices,dists, r*r, flann::SearchParams() );
	printf("done (%g seconds)", stop_timer());

	start_timer("verifying results...");
	for( int i=0; i<query.rows; i++ )
	{
		for( int j=0; j<data.rows; j++ )
		{
			// for each pair of query and data points: either the distance between them
			// is smaller than r AND the point is in the result set, or 
			// the distance is larger and it is not.
			float dist = 0;
			for( int k=0; k<3; k++ )
				dist += (query[i][k]-data[j][k])*(query[i][k]-data[j][k]);
			if( dist < r*r )
			{
				EXPECT_TRUE( std::find( indices[i], indices[i]+max_neighbors, j )!=indices[i]+max_neighbors );
			}
			else
			{
				EXPECT_TRUE( std::find( indices[i], indices[i]+max_neighbors, j )==indices[i]+max_neighbors );
			}
		}
	}
	printf("done (%g seconds)\n", stop_timer());
	delete []counts.ptr();
	delete []indices.ptr();
	delete []dists.ptr();
}

TEST_F(Flann_3D, TestRadiusSearch)
{
    flann::Index<L2_Simple<float> > index(data, flann::KDTreeCuda3dIndexParams());
    start_timer("Building kd-tree index...");
    index.buildIndex();
    printf("done (%g seconds)\n", stop_timer());
	float r = 0.02;
	std::vector< std::vector<int> > indices;
	std::vector< std::vector<float> > dists;
	start_timer("Radius search, r=0.02...");
	index.radiusSearch( query, indices,dists, r*r, flann::SearchParams() );
	printf("done (%g seconds)\n", stop_timer());
	
	start_timer("verifying results...");
	for( int i=0; i<query.rows; i++ )
	{
		for( int j=0; j<data.rows; j++ )
		{
			float dist = 0;
			for( int k=0; k<3; k++ )
				dist += (query[i][k]-data[j][k])*(query[i][k]-data[j][k]);
			if( dist < r*r )
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )!=indices[i].end() );
			}
			else
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )==indices[i].end() );
			}
		}
	}
	printf("done (%g seconds)\n", stop_timer());
	
	r=0.01;
	start_timer("Radius search, r=0.01");
	index.radiusSearch( query, indices,dists, r*r, flann::SearchParams() );
	printf("done (%g seconds)\n", stop_timer());
	
	start_timer("verifying results...");
	for( int i=0; i<query.rows; i++ )
	{
		for( int j=0; j<data.rows; j++ )
		{
			float dist = 0;
			for( int k=0; k<3; k++ )
				dist += (query[i][k]-data[j][k])*(query[i][k]-data[j][k]);
			if( dist < r*r )
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )!=indices[i].end() );
			}
			else
			{
				EXPECT_TRUE( std::find( indices[i].begin(), indices[i].end(), j )==indices[i].end() );
			}
		}
	}
	printf("done (%g seconds)\n", stop_timer());
}

int main(int argc, char** argv)
{
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
